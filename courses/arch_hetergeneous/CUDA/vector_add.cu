#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>


// The number of threads per blocks in the kernel
// (if we define it here, then we can use its value in the kernel,
//  for example to statically declare an array in shared memory)
const int threads_per_block = 256;


// Forward function declarations
float *GPU_add_vectors(float *A, float *B, int N);
float *CPU_add_vectors(float *A, float *B, int N);
float *get_random_vector(int N);
long long start_timer();
long long stop_timer(long long start_time, char *name);
void die(char *message);


int main(int argc, char **argv) {
	// Seed the random generator (use a constant here for repeatable results)
	srand(10);

	// Determine the vector length
	int N = 100000;  // default value
	if (argc > 1) N = atoi(argv[1]); // user-specified value

	// Generate two random vectors
	long long vector_start_time = start_timer();
	float *A = get_random_vector(N);
	float *B = get_random_vector(N);
	stop_timer(vector_start_time, "Vector generation");
	
	// Compute their sum on the GPU
	long long GPU_start_time = start_timer();
	float *C_GPU = GPU_add_vectors(A, B, N);
	long long GPU_time = stop_timer(GPU_start_time, "\t            Total");
	
	// Compute their sum on the CPU
	long long CPU_start_time = start_timer();
	float *C_CPU = CPU_add_vectors(A, B, N);
	long long CPU_time = stop_timer(CPU_start_time, "\nCPU");
	
	// Compute the speedup or slowdown
	if (GPU_time > CPU_time) printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
	else                     printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);
	
	// Check the correctness of the GPU results
	int num_wrong = 0;
	for (int i = 0; i < N; i++) {
		if (fabs(C_CPU[i] - C_GPU[i]) > 0.000001) num_wrong++;
	}
	
	// Report the correctness results
	if (num_wrong) printf("\n%d / %d values incorrect\n", num_wrong, N);
	else           printf("\nAll values correct\n");

}


// A GPU kernel that computes the vector sum A + B
// (each thread computes a single value of the result)
__global__ void add_vectors_kernel(float *A, float *B, float *C, int N) {
	// Determine which element this thread is computing
	int block_id = blockIdx.x + gridDim.x * blockIdx.y;
	int thread_id = blockDim.x * block_id + threadIdx.x;
	
	// Compute a single element of the result vector (if the element is valid)
	if (thread_id < N) C[thread_id] = A[thread_id] + B[thread_id];
}


// Returns the vector sum A + B (computed on the GPU)
float *GPU_add_vectors(float *A_CPU, float *B_CPU, int N) {
	
	long long memory_start_time = start_timer();

	// Allocate GPU memory for the inputs and the result
	int vector_size = N * sizeof(float);
	float *A_GPU, *B_GPU, *C_GPU;
	if (hipMalloc((void **) &A_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &B_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	if (hipMalloc((void **) &C_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
	
	// Transfer the input vectors to GPU memory
	hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
	
	stop_timer(memory_start_time, "\nGPU:\t  Transfer to GPU");
	
	// Determine the number of thread blocks in the x- and y-dimension
	int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
	int max_blocks_per_dimension = 65535;
	int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
	int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
	dim3 grid_size(num_blocks_x, num_blocks_y, 1);
	
	// Execute the kernel to compute the vector sum on the GPU
	long long kernel_start_time = start_timer();
	add_vectors_kernel <<< grid_size , threads_per_block >>> (A_GPU, B_GPU, C_GPU, N);
	hipDeviceSynchronize();  // this is only needed for timing purposes
	stop_timer(kernel_start_time, "\t Kernel execution");
	
	// Check for kernel errors
	hipError_t error = hipGetLastError();
	if (error) {
		char message[256];
		sprintf(message, "CUDA error: %s", hipGetErrorString(error));
		die(message);
	}
	
	// Allocate CPU memory for the result
	float *C_CPU = (float *) malloc(vector_size);
	if (C_CPU == NULL) die("Error allocating CPU memory");
	
	// Transfer the result from the GPU to the CPU
	memory_start_time = start_timer();
	hipMemcpy(C_CPU, C_GPU, vector_size, hipMemcpyDeviceToHost);
	stop_timer(memory_start_time, "\tTransfer from GPU");
	
	// Free the GPU memory
	hipFree(A_GPU);
	hipFree(B_GPU);
	hipFree(C_GPU);
	
	return C_CPU;
}


// Returns the vector sum A + B
float *CPU_add_vectors(float *A, float *B, int N) {	
	// Allocate memory for the result
	float *C = (float *) malloc(N * sizeof(float));
	if (C == NULL) die("Error allocating CPU memory");

	// Compute the sum;
	for (int i = 0; i < N; i++) C[i] = A[i] + B[i];
	
	// Return the result
	return C;
}


// Returns a randomized vector containing N elements
float *get_random_vector(int N) {
	if (N < 1) die("Number of elements must be greater than zero");
	
	// Allocate memory for the vector
	float *V = (float *) malloc(N * sizeof(float));
	if (V == NULL) die("Error allocating CPU memory");
	
	// Populate the vector with random numbers
	for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
	// Return the randomized vector
	return V;
}


// Returns the current time in microseconds
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, char *name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}


// Prints the specified message and quits
void die(char *message) {
	printf("%s\n", message);
	exit(1);
}
