#include "hip/hip_runtime.h"
// -*- Mode: C++ ; c-file-style:"stroustrup"; indent-tabs-mode:nil; -*-

#include "cell.h"
#include <stdio.h>
#include <stdlib.h>
#include "../Model.cu"
#ifndef WIN32
#include <sys/time.h>
#else
#include<time.h>
#endif

// The size of the tile is calculated at compile time by the SL processor.
// But the data array is statically sized.
// So, make these are big as they can get.
// Changed to be large enough for fermi
// (int)cube_rt(1024) = 10
#define TILE_WIDTH  10
#define TILE_HEIGHT 10
#define TILE_DEPTH  10

/**
 * Block of memory shared by threads working on a single tile.
 * Contains all necessary cell values and edge values from the
 * previous iteration.
 */
__shared__ DTYPE shmem[TILE_DEPTH][TILE_HEIGHT][TILE_WIDTH];


/**
 * Store data between calls to SetData() and run().
 * This is basically a hack.
 */
static DTYPE *global_ro_data = NULL;

/**
 * this depends on all blocks being the same size
 */
static DTYPE *device_input = NULL, *device_output = NULL;


__device__ DTYPE get(int x, int y, int z) {
  return shmem[threadIdx.z + z][threadIdx.y + y][threadIdx.x + x];
}

// Macro to read global read only data from within CellValue code.
#define read(offset)(ro_data[offset])

__device__ DTYPE CellValue(dim3 input_size, int x, int y, int z, DTYPE *ro_data
        , int bornMin, int bornMax, int dieMin, int dieMax) {
  int orig = get(0, 0, 0);
  int sum = 0;
  int i, j, k;
  for (i = -1; i <= 1; i++)
    for (j = -1; j <= 1; j++)
      for (k = -1; k <= 1; k++)
        sum += get(i, j, k);
  sum -= orig;
  int retval;
  if (orig > 0 && (sum <= dieMax || sum >= dieMin))
    retval = 0;
  else if (orig == 0 && (sum >= bornMin && sum <= bornMax))
    retval = 1;
  else
    retval = orig;
  return retval;
}

/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernel(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernel(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside;
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;
  inside = ((x == ex) && (y == ey) && (z == ez));
  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}


/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernelOuter(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernelOuter(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside,
          in_ghost_zone(-1);
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;
  
  inside = ((x == ex) && (y == ey) && (z == ez));
  
  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;
  
  const int kGhostZ = kPyramidHeight * stencil_size.z;
  const int kGhostY = kPyramidHeight * stencil_size.y;
  const int kGhostX = kPyramidHeight * stencil_size.x;
  // front and back
  in_ghost_zone = ((ez < kGhostZ) || (ez > input_size.z - kGhostZ));
  // top and bottom
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= input_size.y - kGhostY) && (ey < input_size.y - border.y) &&
          (ex >= border.x) && (ex < input_size.x - border.x));
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= border.y) && (ey < kGhostY) &&
          (ex >= border.x) && (ex < input_size.x - border.x));
  // left and right
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= border.x) && (ex < kGhostX));
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= input_size.x - kGhostX) && (ex < input_size.x - border.x));
  
  inside = inside && in_ghost_zone;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}
/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernelInner(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernelInner(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside,
          in_inner_zone(-1);
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;
  
  inside = ((x == ex) && (y == ey) && (z == ez));
  
  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;
  
  const int kGhostZ = kPyramidHeight * stencil_size.z;
  const int kGhostY = kPyramidHeight * stencil_size.y;
  const int kGhostX = kPyramidHeight * stencil_size.x;
  // front and back
  in_inner_zone = (ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= kGhostX) && (ex < input_size.x - kGhostX);
  inside = inside && in_inner_zone;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCell(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations,
        const int kPyramidHeight, int bornMin, int bornMax, int dieMin,
        int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }
  //hipMemset(static_cast<void*> (device_output), newValue, num_bytes);
  hipMemcpy(device_input, host_data, num_bytes, hipMemcpyHostToDevice);

  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernel";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernel <<<grid_dims, tile_size>>>( input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);

  // Device to host
  hipMemcpy(host_data, device_input, num_bytes, hipMemcpyDeviceToHost);

  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCellInner(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations,
        const int kPyramidHeight, int bornMin, int bornMax, int dieMin,
        int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }
  //hipMemset(static_cast<void*> (device_output), newValue, num_bytes);
  hipMemcpy(device_input, host_data, num_bytes, hipMemcpyHostToDevice);

  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernelInner";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernelInner <<<grid_dims, tile_size>>>( input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);

  // Device to host
  hipMemcpy(host_data, device_input, num_bytes, hipMemcpyDeviceToHost);

  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCellOuter(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations,
        const int kPyramidHeight, int bornMin, int bornMax, int dieMin,
        int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }
  //hipMemset(static_cast<void*> (device_output), newValue, num_bytes);
  hipMemcpy(device_input, host_data, num_bytes, hipMemcpyHostToDevice);

  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernelOuter";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernelOuter <<<grid_dims, tile_size>>>( input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);

  // Device to host
  hipMemcpy(host_data, device_input, num_bytes, hipMemcpyDeviceToHost);

  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

void runCellCleanup() {
  if (device_input != NULL && device_output != NULL) {
    hipFree(device_input);
    device_input = NULL;
    hipFree(device_output);
    device_output = NULL;
  }
}

/**
 * Store unnamed data on device.
 */
void runCellSetData(DTYPE *host_data, int num_elements) {
  int num_bytes = sizeof (DTYPE) * num_elements;
  hipMalloc((void **) &global_ro_data, num_bytes);
  hipMemcpy(global_ro_data, host_data, num_bytes, hipMemcpyHostToDevice);
}
