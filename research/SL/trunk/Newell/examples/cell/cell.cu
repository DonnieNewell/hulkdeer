#include "hip/hip_runtime.h"
// -*- Mode: C++ ; c-file-style:"stroustrup"; indent-tabs-mode:nil; -*-

#include "./cell.h"
#include "../Model.cu"
#include "../comm.h"
#include <stdio.h>
#include <stdlib.h>
#ifndef WIN32
#include <sys/time.h>
#else
#include<time.h>
#endif

static double gpu_memcpy_time = 0.0;

void copyDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size, dim3 border,
        DTYPE* dest_data);
void copyOuterDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size,
        dim3 border, DTYPE* dest_data);
void copyInnerDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size,
        dim3 border, DTYPE* dest_data);
//double secondsElapsed(struct timeval start, struct timeval stop);

// The size of the tile is calculated at compile time by the SL processor.
// But the data array is statically sized.
// So, make these are big as they can get.
// Changed to be large enough for fermi
// (int)cube_rt(1024) = 10
#define TILE_WIDTH  10
#define TILE_HEIGHT 10
#define TILE_DEPTH  10

/**
 * Block of memory shared by threads working on a single tile.
 * Contains all necessary cell values and edge values from the
 * previous iteration.
 */
__shared__ DTYPE shmem[TILE_DEPTH][TILE_HEIGHT][TILE_WIDTH];


/**
 * Store data between calls to SetData() and run().
 * This is basically a hack.
 */
static DTYPE *global_ro_data = NULL;

/**
 * this depends on all blocks being the same size
 */
static DTYPE *device_input = NULL, *device_output = NULL;

double getGpuMemcpyTime() {
  return gpu_memcpy_time;
}

__device__ DTYPE get(int x, int y, int z) {
  return shmem[threadIdx.z + z][threadIdx.y + y][threadIdx.x + x];
}

// Macro to read global read only data from within CellValue code.
#define read(offset)(ro_data[offset])

__device__ DTYPE CellValue(dim3 input_size, int x, int y, int z, DTYPE *ro_data
        , int bornMin, int bornMax, int dieMin, int dieMax) {
  int orig = get(0, 0, 0);
  int sum = 0;
  int i, j, k;
  for (i = -1; i <= 1; i++)
    for (j = -1; j <= 1; j++)
      for (k = -1; k <= 1; k++)
        sum += get(i, j, k);
  sum -= orig;
  int retval;
  if (orig > 0 && (sum <= dieMax || sum >= dieMin))
    retval = 0;
  else if (orig == 0 && (sum >= bornMin && sum <= bornMax))
    retval = 1;
  else
    retval = orig;
  return retval;
}

/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernel(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernel(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside;
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;
  inside = ((x == ex) && (y == ey) && (z == ez));
  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}


/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernelOuter(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernelOuter(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside,
          in_ghost_zone(-1);
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;

  inside = ((x == ex) && (y == ey) && (z == ez));

  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;

  const int kGhostZ = kPyramidHeight * stencil_size.z;
  const int kGhostY = kPyramidHeight * stencil_size.y;
  const int kGhostX = kPyramidHeight * stencil_size.x;
  // front and back
  in_ghost_zone = ((ez < kGhostZ) || (ez > input_size.z - kGhostZ));
#ifndef SLAB
  // top and bottom
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= input_size.y - kGhostY) && (ey < input_size.y - border.y) &&
          (ex >= border.x) && (ex < input_size.x - border.x));
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= border.y) && (ey < kGhostY) &&
          (ex >= border.x) && (ex < input_size.x - border.x));
  // left and right
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= border.x) && (ex < kGhostX));
  in_ghost_zone = in_ghost_zone || ((ez >= kGhostZ) && (ez < input_size.z - kGhostZ) &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= input_size.x - kGhostX) && (ex < input_size.x - border.x));
#endif
  inside = inside && in_ghost_zone;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}
/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the
//  kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only
//  run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
  void runCellKernelInner(dim3 input_size, dim3 stencil_size,
          DTYPE *input, DTYPE *output, int kPyramidHeight,
          DTYPE *ro_data
          , int bornMin, int bornMax, int dieMin, int dieMax);
}

__global__
void runCellKernelInner(dim3 input_size, dim3 stencil_size, DTYPE *input,
        DTYPE *output, int kPyramidHeight, DTYPE *ro_data, int bornMin,
        int bornMax, int dieMin, int dieMax) {
  dim3 border;
  int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside,
          in_inner_zone(-1);
  DTYPE value;

  // (bx, by, bz) is the location in the input of the top left of this block.
  border.x = kPyramidHeight * stencil_size.x;
  border.y = kPyramidHeight * stencil_size.y;
  border.z = kPyramidHeight * stencil_size.z;
  bx = blockIdx.x * (blockDim.x - 2 * border.x) - border.x;
  // These changed by Greg Faust to fix the fact that
  //     grids in CUDA cannot have 3 dimensions.
  // This parallels the same fix Jiayuan Meng used in his code for this issue.
  // UPDATE:(Donnie) There was an error in original version using the
  //  blockdim.x to get the y and z block ID's, this was changed to use
  //  gridDim.x, because blockDim gives the number of threads in a given
  //  direction, while gridDim gives the number of blocks.
  // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
  // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
  int BS = blockDim.x;
  by = (blockIdx.y / gridDim.x) * (BS - 2 * border.y) - border.y;
  bz = (blockIdx.y % gridDim.x) * (BS - 2 * border.z) - border.z;

  // (x, y, z) is the location in the input of this thread.
  tx = threadIdx.x;
  ty = threadIdx.y;
  tz = threadIdx.z;
  x = bx + tx;
  y = by + ty;
  z = bz + tz;

  // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
  // UPDATE:(Donnie) Changed this block to ensure that we only calculate
  //    stencil values for the cells that will have all valid cells in the
  //    stencil calculation.
  ex = x;
  ey = y;
  ez = z;
  const int kValidIndex = 0;
  if (ex < kValidIndex) ex = kValidIndex;
  if (ey < kValidIndex) ey = kValidIndex;
  if (ez < kValidIndex) ez = kValidIndex;
  if (ex >= input_size.x) ex = input_size.x - 1;
  if (ey >= input_size.y) ey = input_size.y - 1;
  if (ez >= input_size.z) ez = input_size.z - 1;

  inside = ((x == ex) && (y == ey) && (z == ez));

  // Get current cell value or edge value.
  //uidx = ez + input_size.y * (ey * input_size.x + ex);
  uidx = ex + input_size.x * (ey + ez * input_size.y);
  value = input[uidx];

  // Store value in shared memory for stencil calculations, and go.
  shmem[tz][ty][tx] = value;

  const int kGhostZ = kPyramidHeight * stencil_size.z;
  const int kGhostY = kPyramidHeight * stencil_size.y;
  const int kGhostX = kPyramidHeight * stencil_size.x;

  // front and back
  in_inner_zone = (ez >= kGhostZ) && (ez < input_size.z - kGhostZ);
#ifndef SLAB
  in_inner_zone = in_inner_zone &&
          (ey >= kGhostY) && (ey < input_size.y - kGhostY) &&
          (ex >= kGhostX) && (ex < input_size.x - kGhostX);
#endif
  
  inside = inside && in_inner_zone;
  iter = 0;
  border.x = border.y = border.z = 0;
  while (true) {
    __syncthreads();
    iter++;
    if (inside) {
      border.x += stencil_size.x;
      border.y += stencil_size.y;
      border.z += stencil_size.z;
      inside = ((tx >= border.x) && (tx < blockDim.x - border.x) &&
              (ty >= border.y) && (ty < blockDim.y - border.y) &&
              (tz >= border.z) && (tz < blockDim.z - border.z));
    }
    if (inside) {
      value = CellValue(input_size, x, y, z, ro_data, bornMin, bornMax,
              dieMin, dieMax);
    }
    if (iter >= kPyramidHeight) {
      if (inside) {
        output[uidx] = value;
      }
      break;
    }
    __syncthreads();
    shmem[tz][ty][tx] = value;
  }
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCell(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations,
        const int kPyramidHeight, int bornMin, int bornMax, int dieMin,
        int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }

  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernel";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  //printf("tile_size(%d, %d, %d)\n", tile_size.x, tile_size.y, tile_size.z);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  //hipMemset(static_cast<void*> (device_output), newValue, num_bytes);
  copyDeviceData(host_data, hipMemcpyHostToDevice, input_size, border, device_input);
  gettimeofday(&end, NULL);
  gpu_memcpy_time += secondsElapsed(start, end);

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernel <<<grid_dims, tile_size >>>(input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);

  // Device to host
  gettimeofday(&start, NULL);
  copyDeviceData(device_input, hipMemcpyDeviceToHost, input_size, border, host_data);
  gettimeofday(&end, NULL);
  gpu_memcpy_time += secondsElapsed(start, end);

  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCellInner(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations,
        const int kPyramidHeight, int bornMin, int bornMax, int dieMin,
        int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }


  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernelInner";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  copyInnerDeviceData(host_data, hipMemcpyHostToDevice, input_size, border,
          device_input);
  gettimeofday(&end, NULL);
  
  gpu_memcpy_time += secondsElapsed(start, end);

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernelInner << <grid_dims, tile_size >> >(input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);

  // Device to host
  gettimeofday(&start, NULL);
  copyInnerDeviceData(device_input, hipMemcpyDeviceToHost, input_size, border,
          host_data);
  gettimeofday(&end, NULL);
  
  gpu_memcpy_time += secondsElapsed(start, end);
  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

/**
 * Function exported to do the entire stencil computation.
 */
void runCellOuter(DTYPE *host_data, int x_max, int y_max, int z_max,
        int iterations, const int kPyramidHeight, int bornMin, int bornMax,
        int dieMin, int dieMax, int device) {
  // User-specific parameters
  dim3 input_size(x_max, y_max, z_max);
  dim3 stencil_size(1, 1, 1);
  //use the appropriate device
  int curr_device = -1;
  struct timeval start, end;

  hipGetDevice(&curr_device);
  if (curr_device != device) {
    //changing devices, so we need to deallocate previous input/output buffers
    runCellCleanup();
    hipError_t err = hipSetDevice(device);
    if (hipSuccess != err) {
      fprintf(stderr, "runCell(): couldn't select GPU index:%d.\nERROR: %s\n",
              device, hipGetErrorString(err));
      return;
    }
  }

  // Allocate CUDA arrays in device memory 
  int num_bytes = input_size.x * input_size.y * input_size.z * sizeof (DTYPE);
  if (NULL == device_input && NULL == device_output) {
    //fprintf(stderr, "allocating gpu memory.\n");
    hipMalloc((void**) &device_output, num_bytes);
    hipMalloc((void**) &device_input, num_bytes);
  }
  // Setup the structure that holds parameters for the application.
  // And from that, get the block size.
  char* KernelName = "runCellKernelOuter";
  dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations,
          sizeof (DTYPE), KernelName);
  dim3 border,
          tile_data_size,
          grid_dims;

  // And use the result to calculate various sizes.
  filldim3(&border,
          kPyramidHeight * stencil_size.x,
          kPyramidHeight * stencil_size.y,
          kPyramidHeight * stencil_size.z);
  filldim3(&tile_data_size,
          tile_size.x - 2 * border.x,
          tile_size.y - 2 * border.y,
          tile_size.z - 2 * border.z);
  filldim3(&grid_dims,
          div_ceil(input_size.x, tile_data_size.x),
          //      div_ceil(input_size.y, tile_data_size.y),
          //      div_ceil(input_size.z, tile_data_size.z));
          div_ceil(input_size.y, tile_data_size.y) *
          div_ceil(input_size.z, tile_data_size.z)); //*/

  gettimeofday(&start, NULL);
  copyOuterDeviceData(host_data, hipMemcpyHostToDevice, input_size, border,
          device_input);
  gettimeofday(&end, NULL);
  
  gpu_memcpy_time += secondsElapsed(start, end);

  gettimeofday(&start, NULL);
  // Run computation
  int tmp_pyramid_height = kPyramidHeight;
  for (int iter = 0; iter < iterations; iter += kPyramidHeight) {
    if (iter + kPyramidHeight > iterations)
      tmp_pyramid_height = iterations - iter;
    runCellKernelOuter << <grid_dims, tile_size >> >(input_size, stencil_size,
            device_input, device_output, tmp_pyramid_height, global_ro_data,
            bornMin, bornMax, dieMin, dieMax);

    DTYPE *temp = device_input;
    device_input = device_output;
    device_output = temp;
  }
  gettimeofday(&end, NULL);
  
// Device to host
  gettimeofday(&start, NULL);
  copyOuterDeviceData(device_input, hipMemcpyDeviceToHost, input_size, border,
          host_data);
  gettimeofday(&end, NULL);
  gpu_memcpy_time += secondsElapsed(start, end);

  if (global_ro_data != NULL) {
    hipFree(global_ro_data);
    global_ro_data = NULL;
  }

  //disposeSAProps(SAPs);
  SAPs = NULL;
}

void runCellCleanup() {
  if (device_input != NULL && device_output != NULL) {
    hipFree(device_input);
    device_input = NULL;
    hipFree(device_output);
    device_output = NULL;
  }
}

/**
 * Store unnamed data on device.
 */
void runCellSetData(DTYPE *host_data, int num_elements) {
  int num_bytes = sizeof (DTYPE) * num_elements;
  hipMalloc((void **) &global_ro_data, num_bytes);
  hipMemcpy(global_ro_data, host_data, num_bytes, hipMemcpyHostToDevice);
}

void copyDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size, dim3 border,
        DTYPE* dest_data) {
  const int kNumElements = size.x * size.y * size.z;
  void* dest = static_cast<void*> (dest_data);
  void* src = static_cast<void*> (src_data);
  size_t num_bytes = kNumElements * sizeof (src_data[0]);
  hipMemcpy(dest, src, num_bytes, kind);
}

void copyOuterDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size,
        dim3 border, DTYPE* dest_data) {
  // front
  DTYPE *dest_start = dest_data;
  DTYPE *src_start = src_data;
  int num_elements = border.z * size.y * size.x;
  size_t num_bytes = num_elements * sizeof (src_data[0]);
  void* dest = static_cast<void*> (dest_start);
  void* src = static_cast<void*> (src_start);
  hipMemcpy(dest, src, num_bytes, kind);

  // back
  int offset = (size.z - border.z) * size.y * size.x;
  src_start = src_data + offset;
  dest_start = dest_data + offset;
  dest = static_cast<void*> (dest_start);
  src = static_cast<void*> (src_start);
  hipMemcpy(dest, src, num_bytes, kind);

#ifndef SLAB
  // top
  num_elements = size.x;
  num_bytes = num_elements * sizeof (src_data[0]);
  int pitch = size.x * size.y * sizeof (src_data[0]);
  offset = border.z * size.y * size.x + (size.y - border.y) * size.x;
  src_start = src_data + offset;
  dest_start = dest_data + offset;
  dest = static_cast<void*> (dest_start);
  src = static_cast<void*> (src_start);
  hipMemcpy2D(dest, pitch, src, pitch, num_bytes, border.y, kind);

  // bottom
  offset = border.z * size.y * size.x;
  src_start = src_data + offset;
  dest_start = dest_data + offset;
  dest = static_cast<void*> (dest_start);
  src = static_cast<void*> (src_start);
  hipMemcpy2D(dest, pitch, src, pitch, num_bytes, border.y, kind);

  // left
  num_elements = border.x;
  num_bytes = num_elements * sizeof (src_data[0]);
  pitch = size.x * sizeof (src_data[0]);
  for (int i = border.z; i < size.z - border.z; ++i) {
    offset = i * size.y * size.x + border.y * size.x;
    src_start = src_data + offset;
    dest_start = dest_data + offset;
    dest = static_cast<void*> (dest_start);
    src = static_cast<void*> (src_start);
    hipMemcpy2D(dest, pitch, src, pitch, num_bytes, size.y - 2 * border.y,
            kind);
  }

  // right
  for (int i = border.z; i < size.z - border.z; ++i) {
    offset = i * size.y * size.x + border.y * size.x + size.x - border.x;
    src_start = src_data + offset;
    dest_start = dest_data + offset;
    dest = static_cast<void*> (dest_start);
    src = static_cast<void*> (src_start);
    hipMemcpy2D(dest, pitch, src, pitch, num_bytes, size.y - 2 * border.y,
            kind);
  }
#endif
}

void copyInnerDeviceData(DTYPE* src_data, enum hipMemcpyKind kind, dim3 size,
        dim3 border, DTYPE* dest_data) {
#ifndef SLAB
  int num_elements = size.x - 2 * border.x;
  int num_bytes = num_elements * sizeof (src_data[0]);
  for (int i = border.z; i < size.z - border.z; ++i) {
    int offset = i * size.y * size.x + border.y * size.x + border.x;
    DTYPE* src_start = src_data + offset;
    DTYPE* dest_start = dest_data + offset;
    void* dest = static_cast<void*> (dest_start);
    void* src = static_cast<void*> (src_start);
    int pitch = size.x * sizeof (src_data[0]);
    hipMemcpy2D(dest, pitch, src, pitch, num_bytes, size.y - 2 * border.y,
            kind);
  }
#else
  int num_elements = (size.z - 2 * border.z) * size.y * size.x;
  int num_bytes = num_elements * sizeof (src_data[0]);
  int offset = border.z * size.y * size.x;
  DTYPE* src_start = src_data + offset;
  DTYPE* dest_start = dest_data + offset;
  void* dest = static_cast<void*> (dest_start);
  void* src = static_cast<void*> (src_start);
  int pitch = size.x * sizeof (src_data[0]);
  hipMemcpy(dest, src, num_bytes, kind);
#endif
}
