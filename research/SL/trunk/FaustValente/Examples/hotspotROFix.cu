#include "hip/hip_runtime.h"
// -*- Mode: C++ ; c-file-style:"stroustrup"; indent-tabs-mode:nil; -*-

#include <stdio.h>
#include <stdlib.h>
#include "Model.cu"

#define DTYPE float

// The size of the tile is calculated at compile time by the SL processor.
// But the data array is statically sized.
// So, make these are big as they can get.
// (int)sqrt(512) = 22
#define TILE_WIDTH  32
#define TILE_HEIGHT 32

/**
 * Block of memory shared by threads working on a single tile.
 * Contains all necessary cell values and edge values from the
 * previous iteration.
 */
__shared__ DTYPE shmem[TILE_HEIGHT][TILE_WIDTH];

__device__ DTYPE get(int x, int y)
{
    return shmem[threadIdx.y+y][threadIdx.x+x];
}

/**
 * Store data between calls to SetData() and run().
 * This is basically a hack.
 */
static DTYPE *global_ro_data = NULL;

// Macro to read global read only data from within CellValue code.
#define read(offset)(ro_data[offset])

__device__ DTYPE CellValue(dim3 input_size, int x, int y, DTYPE pvalue //*ro_data
                           , float step_div_Cap, float Rx, float Ry, float Rz)
{
    float value, term1, term2, term3, sum;
    // pvalue = read(y * input_size.x + x);
    value = get(0, 0);
    term1 = (get(0, 1) + get(0, -1) - value - value) / Ry;
    term2 = (get(1, 0) + get(-1, 0) - value - value) / Rx;
    term3 = (80.0 - value) / Rz;
    sum = pvalue + term1 + term2 + term3;
    return(value + step_div_Cap * sum);
}


/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
void runHotspotKernel(dim3 input_size, dim3 stencil_size,
                          DTYPE *input, DTYPE *output, int pyramid_height,
                          DTYPE *ro_data
                          , float step_div_Cap, float Rx, float Ry, float Rz);
    }

__global__
void runHotspotKernel(dim3 input_size, dim3 stencil_size,
                          DTYPE *input, DTYPE *output, int pyramid_height,
                          DTYPE *ro_data
                          , float step_div_Cap, float Rx, float Ry, float Rz)
{
    dim3 border;
    int bx, by, tx, ty, x, y, ex, ey, uidx, iter, inside;
    DTYPE value;

    // (bx, by) is the location in the input of the top left of this block.
    border.x = pyramid_height * stencil_size.x;
    border.y = pyramid_height * stencil_size.y;
    bx = blockIdx.x * (blockDim.x - 2*border.x) - border.x;
    by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
    // (x, y) is the location in the input of this thread.
    tx = threadIdx.x;
    ty = threadIdx.y;
    x = bx + tx;
    y = by + ty;

    // (ex, ey) = (x, y) pushed into the boundaries of the input.
    ex = x;
    ey = y;
    if (ex < 0) ex = 0;
    if (ey < 0) ey = 0;
    if (ex >= input_size.x) ex = input_size.x-1;
    if (ey >= input_size.y) ey = input_size.y-1;

    // Get current cell value or edge value.
    uidx = ey * input_size.x + ex;
    value = input[uidx];
    inside = ((x == ex) && (y == ey));

    // Store value in shared memory for stencil calculations, and go.
    DTYPE pvalue = 0;
    // This works worse for PH=1, but better otherwise.
    if (inside) pvalue = ro_data[y * input_size.x + x];
    shmem[ty][tx] = value;
    iter = 0;
    border.x = border.y = 0;
    while (true)
    {
        __syncthreads();
        iter++;
        if (inside)
        {
            border.x += stencil_size.x;
            border.y += stencil_size.y;
            inside = ((tx >= border.x) && (tx < blockDim.x-border.x) &&
                      (ty >= border.y) && (ty < blockDim.y-border.y));
        }
        if (inside)
        {
            // This works better for PH=1, but worse otherwise.
            // if (iter == 1) pvalue = ro_data[y * input_size.x + x];
            value = CellValue(input_size, x, y, pvalue
                              , step_div_Cap, Rx, Ry, Rz);
        }
        if (iter >= pyramid_height)
        {
            if (inside)
                output[uidx] = value;
            break;
        }
        __syncthreads();
        shmem[ty][tx] = value;
    }
}

/**
 * Function exported to do the entire stencil computation.
 */
void runHotspot(DTYPE *host_data, int x_max, int y_max, int iterations
                    , float step_div_Cap, float Rx, float Ry, float Rz)
{
    // User-specific parameters
    dim3 input_size(x_max, y_max);
    dim3 stencil_size(1,1);

    // Host to device
    DTYPE *device_input, *device_output;
    int num_bytes = input_size.x * input_size.y * sizeof(DTYPE);
    hipMalloc((void **) &device_input, num_bytes);
    hipMalloc((void **) &device_output, num_bytes);
    hipMemcpy(device_input, host_data, num_bytes, hipMemcpyHostToDevice);

    // Setup the structure that holds parameters for the application.
    // And from that, get the block size.
    char * KernelName = "runHotspotKernel";
    dim3 tile_size = initSAProps(2, input_size, stencil_size, iterations, sizeof(DTYPE), KernelName);

    dim3 border, tile_data_size, grid_dims;

    // Now ready for the training period.
    // Need to get some timings of small kernel runs.
    // TODO It would be faster if these could be 0 and 1 heights instead of 1 and 2.
    int pyramid_height = 2;
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y));
    unsigned int twoIterTime;
    timeInMicroSeconds(twoIterTime, (runHotspotKernel<<< grid_dims, tile_size >>>(
                                    input_size, stencil_size, device_input, device_output,
                                    pyramid_height, global_ro_data
                                    , step_div_Cap, Rx, Ry, Rz)));
    pyramid_height = 1;
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y));
    unsigned int oneIterTime;
    timeInMicroSeconds(oneIterTime, (runHotspotKernel<<< grid_dims, tile_size >>>(
                                    input_size, stencil_size, device_input, device_output,
                                    pyramid_height, global_ro_data
                                    , step_div_Cap, Rx, Ry, Rz)));

#ifdef STATISTICS
/////////////////////////////////////////////////////////////////////////////////////
// Start of code to gather statistics to hone model.  Remove in final version.
////////////////////////////////////////////////////////////////////////////////////

    fprintf(stderr, "***********************************Start of a new Run****************************************\n");
    fprintf(stderr, "Data Size=%d, Tile Size=%d Iteration Count=%d\n", input_size.x, tile_size.x, iterations);

    // Precalculate the pyramid height so we can get stats on the calculated value.
    int calcMinPyramid = calcPyramidHeight(grid_dims, oneIterTime, twoIterTime);
    // Get second best for same reason.
    int secondMinPyramid = getSecond(calcMinPyramid);

    // Gather statistics to help hone model.
    double calcMinTime, secondMinTime;
    double actualMinTime = 1000000000;
    int actualMinPyramid;
    // Now let's just try them all to see what the optimal pyramid height is.
    for (int i=1; i<tile_size.x/2 - border.x; i++)
    {
        int pyramid_height = i;

        // Now we can calculate the other sizes.
        dim3 border(pyramid_height * stencil_size.x,
                    pyramid_height * stencil_size.y);
        dim3 tile_data_size(tile_size.x - 2*border.x,
                            tile_size.y - 2*border.y);
        dim3 grid_dims(div_ceil(input_size.x, tile_data_size.x),
                       div_ceil(input_size.y, tile_data_size.y));

        uint32_t time;
        timeInMicroSeconds(time, (runHotspotKernel<<< grid_dims, tile_size >>>(
                                      input_size, stencil_size, device_input, device_output,
                                      i, global_ro_data
                                      , step_div_Cap, Rx, Ry, Rz)));
        
        double timePer = ((double)time)/i;
        if (i == calcMinPyramid) calcMinTime = timePer;
        if (i == secondMinPyramid) secondMinTime = timePer;
        if (timePer < actualMinTime)
        {
            actualMinPyramid = i;
            actualMinTime = timePer;
        }
        // fprintf(stderr, "Pyramid Height=%d, time=%u, Time per iteration=%f.\n", i, time, ((double)time/i));
    }

    // Now we can output some statistics.
    double firstError = ((1. - (actualMinTime/calcMinTime)) * 100.);
    double secondError = ((1. - (actualMinTime/secondMinTime)) * 100.);
    fprintf(stderr, "Size %d BestHeight %d CalcHeight %d %%Slowdown %4.2f CalcSecond %d %%Slowdown %4.2f MinSlowdown %4.2f\n", 
            input_size.x, actualMinPyramid, calcMinPyramid, firstError, secondMinPyramid, secondError, MIN(firstError, secondError));

/////////////////////////////////////////////////////////////////////////////////////
// End of code to gather statistics to hone model.  Remove in final version.
////////////////////////////////////////////////////////////////////////////////////
#endif

#ifdef STATISTICS

    for (int i=1; i<=tile_size.x/2 - stencil_size.x; i++)
    {
        struct timeval starttime, endtime;
        unsigned int usec2;
        gettimeofday(&starttime, NULL);                                       

        pyramid_height=i;

#else
    // Now we can calculate the pyramid height.
    pyramid_height = calcPyramidHeight(grid_dims, oneIterTime, twoIterTime);
#endif

    // And use the result to calculate various sizes.
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y));

    // Run computation
    for (int iter = 0; iter < iterations; iter += pyramid_height)
    {
        if (iter + pyramid_height > iterations)
            pyramid_height = iterations - iter;
        runHotspotKernel<<< grid_dims, tile_size >>>(
            input_size, stencil_size, device_input, device_output,
            pyramid_height, global_ro_data
            , step_div_Cap, Rx, Ry, Rz);
        DTYPE *temp = device_input;
        device_input = device_output;
        device_output = temp;
    }
#ifdef STATISTICS
    // Synch the threads to make sure everything is done before taking a timing.
    CUDA_SAFE_THREAD_SYNC();
    gettimeofday(&endtime, NULL);                                       
    usec2 = ((endtime.tv_sec - starttime.tv_sec) * 1000000 +             
             (endtime.tv_usec - starttime.tv_usec));                       
    fprintf(stderr, "Actual pyramid=%d, Actual total time=%Lu\n", i, usec2);
    }
#endif

    // Device to host
    hipMemcpy(host_data, device_input, num_bytes, hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    if (global_ro_data != NULL)
    {
        hipFree(global_ro_data);
        global_ro_data = NULL;
    }
}

/**
 * Store unnamed data on device.
 */
void runHotspotSetData(DTYPE *host_data, int num_elements)
{
    int num_bytes = sizeof(DTYPE) * num_elements;
    hipMalloc((void **) &global_ro_data, num_bytes);
    hipMemcpy(global_ro_data, host_data, num_bytes, hipMemcpyHostToDevice);
}
