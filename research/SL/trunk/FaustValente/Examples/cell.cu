#include "hip/hip_runtime.h"
// -*- Mode: C++ ; c-file-style:"stroustrup"; indent-tabs-mode:nil; -*-

#include <stdio.h>
#include <stdlib.h>
#include "Model.cu"

#define DTYPE int

// The size of the tile is calculated at compile time by the SL processor.
// But the data array is statically sized.
// So, make these are big as they can get.
// Changed to be large enough for fermi
// (int)cube_rt(1024) = 10
#define TILE_WIDTH  10
#define TILE_HEIGHT 10
#define TILE_DEPTH  10

/**
 * Block of memory shared by threads working on a single tile.
 * Contains all necessary cell values and edge values from the
 * previous iteration.
 */
__shared__ DTYPE shmem[TILE_DEPTH][TILE_HEIGHT][TILE_WIDTH];

__device__ DTYPE get(int x, int y, int z)
{
    return shmem[threadIdx.z+z][threadIdx.y+y][threadIdx.x+x];
}

// Macro to read global read only data from within CellValue code.
#define read(offset)(ro_data[offset])

__device__ DTYPE CellValue(dim3 input_size, int x, int y, int z, DTYPE *ro_data
                           , int bornMin, int bornMax, int dieMin, int dieMax)
{
    int orig = get(0, 0, 0);
    int sum = 0;
    int i, j, k;
    for (i = -1; i <= 1; i++)
        for (j = -1; j <= 1; j++)
	    for (k = -1; k <= 1; k++)
	    	sum += get(i, j, k);
    sum -= orig;
    int retval;
    if(orig>0 && (sum <= dieMax || sum >= dieMin)) retval = 0;
    else if (orig==0 && (sum >= bornMin && sum <= bornMax)) retval = 1;
    else retval = orig;    
    return (retval);
}

/**
 * Each thread runs this kernel to calculate the value at one particular
 * cell in one particular iteration.
 */

// We need to declare it C style naming.
// This avoids name mangling and allows us to get attributes about the kernel call from Cuda.
// Its possible to do this with a C++ interface, but that will only run on certain devices.
// This technique is older and therefore more reliable across Cuda devices.
extern "C" {
void runCellKernel(dim3 input_size, dim3 stencil_size,
                          DTYPE *input, DTYPE *output, int pyramid_height,
                          DTYPE *ro_data
                          , int bornMin, int bornMax, int dieMin, int dieMax);
    }

__global__
void runCellKernel(dim3 input_size, dim3 stencil_size,
                          DTYPE *input, DTYPE *output, int pyramid_height,
                          DTYPE *ro_data
                          , int bornMin, int bornMax, int dieMin, int dieMax)
{
    dim3 border;
    int bx, by, bz, tx, ty, tz, x, y, z, ex, ey, ez, uidx, iter, inside;
    DTYPE value;

    // (bx, by, bz) is the location in the input of the top left of this block.
    border.x = pyramid_height * stencil_size.x;
    border.y = pyramid_height * stencil_size.y;
    border.z = pyramid_height * stencil_size.z;
    bx = blockIdx.x * (blockDim.x - 2*border.x) - border.x;
    // These changed by Greg Faust to fix the fact that
    //     grids in CUDA cannot have 3 dimensions.
    // This parallels the same fix Jiayuan Meng used in his code for this issue.
    // by = blockIdx.y * (blockDim.y - 2*border.y) - border.y;
    // bz = blockIdx.z * (blockDim.z - 2*border.z) - border.z;
    int BS = blockDim.x;
    by = (blockIdx.y/BS) * (BS - 2*border.y) - border.y;
    bz = (blockIdx.y%BS) * (BS - 2*border.z) - border.z;

    // (x, y, z) is the location in the input of this thread.
    tx = threadIdx.x;
    ty = threadIdx.y;
    tz = threadIdx.z;
    x = bx + tx;
    y = by + ty;
    z = bz + tz;

    // (ex, ey, ez) = (x, y, z) pushed into the boundaries of the input.
    ex = x;
    ey = y;
    ez = z;
    if (ex < 0) ex = 0;
    if (ey < 0) ey = 0;
    if (ez < 0) ez = 0;
    if (ex >= input_size.x) ex = input_size.x-1;
    if (ey >= input_size.y) ey = input_size.y-1;
    if (ez >= input_size.z) ez = input_size.z-1;

    // Get current cell value or edge value.
    // uidx = ez + input_size.y * (ey * input_size.x + ex);
    uidx = ex + input_size.x * (ey + ez * input_size.y);
    value = input[uidx];
    inside = ((x == ex) && (y == ey) && (z == ez));

    // Store value in shared memory for stencil calculations, and go.
    shmem[tz][ty][tx] = value;
    iter = 0;
    border.x = border.y = border.z = 0;
    while (true)
    {
        __syncthreads();
        iter++;
        if (inside)
        {
            border.x += stencil_size.x;
            border.y += stencil_size.y;
            border.z += stencil_size.z;
            inside = ((tx >= border.x) && (tx < blockDim.x-border.x) &&
                      (ty >= border.y) && (ty < blockDim.y-border.y) &&
                      (tz >= border.z) && (tz < blockDim.z-border.z));
        }
        if (inside)
        {
            value = CellValue(input_size, x, y, z, ro_data
                              , bornMin, bornMax, dieMin, dieMax);
        }
        if (iter >= pyramid_height)
        {
            if (inside)
                output[uidx] = value;
            break;
        }
        __syncthreads();

        shmem[tz][ty][tx] = value;
    }
}

/**
 * Store data between calls to SetData() and run().
 * This is basically a hack.
 */
static DTYPE *global_ro_data = NULL;

/**
 * Function exported to do the entire stencil computation.
 */
void runCell(DTYPE *host_data, int x_max, int y_max, int z_max, int iterations
                    , int bornMin, int bornMax, int dieMin, int dieMax)
{
    // User-specific parameters
    dim3 input_size(x_max, y_max, z_max);
    dim3 stencil_size(1,1,1);

    // Host to device
    DTYPE *device_input, *device_output;
    int num_bytes = input_size.x * input_size.y * input_size.z * sizeof(DTYPE);
    hipMalloc((void **) &device_input, num_bytes);
    hipMalloc((void **) &device_output, num_bytes);
    hipMemcpy(device_input, host_data, num_bytes, hipMemcpyHostToDevice);

#ifdef STATISTICS
    struct timeval trainingstarttime, trainingendtime;
    unsigned int trainingusec;
    gettimeofday(&trainingstarttime, NULL);                                       
#endif

    // Setup the structure that holds parameters for the application.
    // And from that, get the block size.
    char * KernelName = "runCellKernel";
    dim3 tile_size = initSAProps(3, input_size, stencil_size, iterations, sizeof(DTYPE), KernelName);

    dim3 border, tile_data_size, grid_dims;

    // Now ready for the training period.
    // Need to get some timings of small kernel runs.
    // TODO It would be faster if these could be 0 and 1 heights instead of 1 and 2.
    int pyramid_height = 2;
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y, pyramid_height * stencil_size.z);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y, tile_size.z - 2*border.z);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y)*div_ceil(input_size.z, tile_data_size.z));
    unsigned int twoIterTime;
    timeInMicroSeconds(twoIterTime, (runCellKernel<<< grid_dims, tile_size >>>(
                                    input_size, stencil_size, device_input, device_output,
                                    pyramid_height, global_ro_data
                                    , bornMin, bornMax, dieMin, dieMax)));
    pyramid_height = 1;
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y, pyramid_height * stencil_size.z);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y, tile_size.z - 2*border.z);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y)*div_ceil(input_size.z, tile_data_size.z));
    unsigned int oneIterTime;
    timeInMicroSeconds(oneIterTime, (runCellKernel<<< grid_dims, tile_size >>>(
                                    input_size, stencil_size, device_input, device_output,
                                    pyramid_height, global_ro_data
                                    , bornMin, bornMax, dieMin, dieMax)));

#ifdef STATISTICS
/////////////////////////////////////////////////////////////////////////////////////
// Start of code to gather statistics to hone model.  Remove in final version.
////////////////////////////////////////////////////////////////////////////////////

    fprintf(stderr, "***********************************Start of a new Run****************************************\n");
    fprintf(stderr, "Data Size=%d, Tile Size=%d Iteration Count=%d\n", input_size.x, tile_size.x, iterations);

    // Precalculate the pyramid height so we can get stats on the calculated value.
    int calcMinPyramid = calcPyramidHeight(grid_dims, oneIterTime, twoIterTime);

    gettimeofday(&trainingendtime, NULL);
    trainingusec = ((trainingendtime.tv_sec - trainingstarttime.tv_sec) * 1000000 +             
                    (trainingendtime.tv_usec - trainingstarttime.tv_usec));                       

    // Get second best for same reason.
    int secondMinPyramid = getSecond(calcMinPyramid);

    // Gather statistics to help hone model.
    double calcMinTime, secondMinTime;
    double actualMinTime = 1000000000;
    int actualMinPyramid;
    // Now let's just try them all to see what the optimal pyramid height is.
    for (int i=1; i<tile_size.x/(2 * stencil_size.x); i++)
    {
        int pyramid_height = i;

        // Now we can calculate the other sizes.
        dim3 border(pyramid_height * stencil_size.x,
                    pyramid_height * stencil_size.y,
                    pyramid_height * stencil_size.z);
        dim3 tile_data_size(tile_size.x - 2*border.x,
                            tile_size.y - 2*border.y,
                            tile_size.z - 2*border.z);
        dim3 grid_dims(div_ceil(input_size.x, tile_data_size.x),
                       div_ceil(input_size.y, tile_data_size.y)*
                       div_ceil(input_size.z, tile_data_size.z));

        uint32_t time;
        timeInMicroSeconds(time, (runCellKernel<<< grid_dims, tile_size >>>(
                                      input_size, stencil_size, device_input, device_output,
                                      i, global_ro_data
                                      , bornMin, bornMax, dieMin, dieMax)));
        
        double timePer = ((double)time)/i;
        if (i == calcMinPyramid) calcMinTime = timePer;
        if (i == secondMinPyramid) secondMinTime = timePer;
        if (timePer < actualMinTime)
        {
            actualMinPyramid = i;
            actualMinTime = timePer;
        }
        // fprintf(stderr, "Pyramid Height=%d, time=%u, Time per iteration=%f.\n", i, time, ((double)time/i));
    }

    // Now we can output some statistics.
    double firstError = ((1. - (actualMinTime/calcMinTime)) * 100.);
    double secondError = ((1. - (actualMinTime/secondMinTime)) * 100.);
    fprintf(stderr, "Size %d BestHeight %d CalcHeight %d %%Slowdown %4.2f CalcSecond %d %%Slowdown %4.2f MinSlowdown %4.2f\n", 
            input_size.x, actualMinPyramid, calcMinPyramid, firstError, secondMinPyramid, secondError, MIN(firstError, secondError));

/////////////////////////////////////////////////////////////////////////////////////
// End of code to gather statistics to hone model.  Remove in final version.
////////////////////////////////////////////////////////////////////////////////////
#endif

#ifdef STATISTICS

    for (int i=1; i<tile_size.x/(2 * stencil_size.x); i++)
    {
        struct timeval starttime, endtime;
        unsigned int usec2;
        gettimeofday(&starttime, NULL);                                       

        pyramid_height=i;

#else
    // Now we can calculate the pyramid height.
    pyramid_height = calcPyramidHeight(grid_dims, oneIterTime, twoIterTime);
#endif

    // And use the result to calculate various sizes.
    filldim3(&border, pyramid_height * stencil_size.x, pyramid_height * stencil_size.y, pyramid_height * stencil_size.z);
    filldim3(&tile_data_size, tile_size.x - 2*border.x, tile_size.y - 2*border.y, tile_size.z - 2*border.z);
    filldim3(&grid_dims, div_ceil(input_size.x, tile_data_size.x), div_ceil(input_size.y, tile_data_size.y)*div_ceil(input_size.z, tile_data_size.z));

    // Run computation
    for (int iter = 0; iter < iterations; iter += pyramid_height)
    {
        if (iter + pyramid_height > iterations)
            pyramid_height = iterations - iter;
        runCellKernel<<< grid_dims, tile_size >>>(
            input_size, stencil_size, device_input, device_output,
            pyramid_height, global_ro_data
            , bornMin, bornMax, dieMin, dieMax);
        DTYPE *temp = device_input;
        device_input = device_output;
        device_output = temp;
    }

#ifdef STATISTICS
    // Synch the threads to make sure everything is done before taking a timing.
    CUDA_SAFE_THREAD_SYNC();
    gettimeofday(&endtime, NULL);                                       
    usec2 = ((endtime.tv_sec - starttime.tv_sec) * 1000000 +             
             (endtime.tv_usec - starttime.tv_usec));                       
    fprintf(stderr, "Actual pyramid=%d, Actual iteration time=%Lu, Actual Total time=%lu\n", i, usec2, usec2+trainingusec);
    }
#endif

    // Device to host
    hipMemcpy(host_data, device_input, num_bytes, hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    if (global_ro_data != NULL)
    {
        hipFree(global_ro_data);
        global_ro_data = NULL;
    }
}

/**
 * Store unnamed data on device.
 */
void runCellSetData(DTYPE *host_data, int num_elements)
{
    int num_bytes = sizeof(DTYPE) * num_elements;
    hipMalloc((void **) &global_ro_data, num_bytes);
    hipMemcpy(global_ro_data, host_data, num_bytes, hipMemcpyHostToDevice);
}
