#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>

#include "timer.h"
//#define BLOCK_SIZE 256
#include "blocksize.h"

#define STR_SIZE 256

#define DEVICE 1

#define HALO 1 // halo width along one direction when advancing to the next iteration

void run(int argc, char** argv);

/* define timer macros */
#define pin_stats_reset()   startCycle()
#define pin_stats_pause(cycles)   stopCycle(cycles)
#define pin_stats_dump(cycles)    printf("timer: %Lu\n", cycles)

int rows, cols;
int* data;
int** wall;
int* result;
#define M_SEED 9
int pyramid_height;

//#define BENCH_PRINT


void
init(int argc, char** argv)
{
	if(argc==4){

		cols = atoi(argv[1]);

		rows = atoi(argv[2]);

                pyramid_height=atoi(argv[3]);
	}else{
                printf("Usage: dynproc row_len col_len pyramid_height\n");
                exit(0);
        }
	data = new int[rows*cols];

	wall = new int*[rows];

	for(int n=0; n<rows; n++)

		wall[n]=data+cols*n;

	result = new int[cols];

	

	int seed = M_SEED;

	srand(seed);



	for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            wall[i][j] = rand() % 10;

        }

    }

#ifdef BENCH_PRINT

    for (int i = 0; i < rows; i++)

    {

        for (int j = 0; j < cols; j++)

        {

            printf("%d ",wall[i][j]) ;

        }

        printf("\n") ;

    }

#endif
}

void 
fatal(char *s)
{
	fprintf(stderr, "error: %s\n", s);

}

#define IN_RANGE(x, min, max)   ((x)>=(min) && (x)<=(max))
#define CLAMP_RANGE(x, min, max) x = (x<(min)) ? min : ((x>(max)) ? max : x )
#define MIN(a, b) ((a)<=(b) ? (a) : (b))

__global__ void dynproc_kernel(
                int iteration, 
                int trpzheight,
                int *gpuWall,
                int *gpuSrc,
                int *gpuResults,
                int cols, 
                int rows,
                int startStep,
                int border)
{

        __shared__ int prev[BLOCK_SIZE];
        __shared__ int result[BLOCK_SIZE];

	int bx = blockIdx.x;
	int tx=threadIdx.x;
	
        // each block finally computes result for a small block
        // after N iterations. 
        // it is the non-overlapping small blocks that cover 
        // all the input data

        // calculate the small block size
	int small_block_cols = BLOCK_SIZE-trpzheight*HALO*2;

        // calculate the boundary for the block according to 
        // the boundary of its small block
        int blkX = small_block_cols*bx-border;

        // calculate the global thread coordination
	int xidx = blkX+tx;
       
        int W = tx-1;
        int E = tx+1;

        W = (W+blkX < 0 ) ? -blkX : W;
        E = (E+blkX >= cols) ? cols-1-blkX : E;
        
        bool isValid = false;

	if(IN_RANGE(xidx, 0, cols-1)){
            prev[tx] = gpuSrc[xidx];
            isValid = true;
	}
        if(!isValid)
            return;
        __syncthreads();

        int i;
        for (i=0; i<iteration ; i++){ 
                  int left = prev[W];
                  int up = prev[tx];
                  int right = prev[E];
                  int shortest = MIN(left, up);
                  shortest = MIN(shortest, right);
                  int index = cols*(startStep+i)+xidx;
                  result[tx] = shortest + gpuWall[index];
	
            __syncthreads();
            if(i==iteration-1)
                break;
            prev[tx]= result[tx];
      }

      // update the global memory
      // after the last iteration, only threads coordinated within the 
      // small block perform the calculation and switch on ``computed''
      if( IN_RANGE(tx, trpzheight, BLOCK_SIZE-trpzheight-1) &&  \
                  isValid){
          gpuResults[xidx]=result[tx];		
      }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols, \
	 int pyramid_height, int blockCols, int borderCols)
{
        dim3 dimBlock(BLOCK_SIZE);
        dim3 dimGrid(blockCols);  
	
        int src = 1, dst = 0;
	for (int t = 0; t < rows-1; t+=pyramid_height) {
            int temp = src;
            src = dst;
            dst = temp;
            dynproc_kernel<<<dimGrid, dimBlock>>>(
                MIN(pyramid_height, rows-t-1), pyramid_height,
                gpuWall, gpuResult[src], gpuResult[dst],
                cols,rows, t, borderCols);
	}
        return dst;
}

int main(int argc, char** argv)
{
    int num_devices;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) hipSetDevice(DEVICE);

    run(argc,argv);

    return EXIT_SUCCESS;
}

void run(int argc, char** argv)
{
    init(argc, argv);

    /* --------------- pyramid parameters --------------- */
    int borderCols = (pyramid_height)*HALO;
    int smallBlockCol = BLOCK_SIZE-(pyramid_height)*HALO*2;
    int blockCols = cols/smallBlockCol+((cols%smallBlockCol==0)?0:1);

    printf("pyramidHeight: %d\ngridSize: [%d]\nborder:[%d]\nblockSize: %d\nblockGrid:[%d]\ntargetBlock:[%d]\n",\
	pyramid_height, cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);
	
    int *gpuWall, *gpuResult[2];
    int size = rows*cols;

    hipMalloc((void**)&gpuResult[0], sizeof(int)*cols);
    hipMalloc((void**)&gpuResult[1], sizeof(int)*cols);
    hipMemcpy(gpuResult[0], data, sizeof(int)*cols, hipMemcpyHostToDevice);
    hipMalloc((void**)&gpuWall, sizeof(int)*(size-cols));
    hipMemcpy(gpuWall, data+cols, sizeof(int)*(size-cols), hipMemcpyHostToDevice);

    unsigned long long cycles;
    pin_stats_reset();

    int final_ret = calc_path(gpuWall, gpuResult, rows, cols, \
	 pyramid_height, blockCols, borderCols);

    hipMemcpy(result, gpuResult[final_ret], sizeof(int)*cols, hipMemcpyDeviceToHost);

    pin_stats_pause(cycles);
    pin_stats_dump(cycles);

#ifdef BENCH_PRINT

    for (int i = 0; i < cols; i++)

            printf("%d ",data[i]) ;

    printf("\n") ;

    for (int i = 0; i < cols; i++)

            printf("%d ",result[i]) ;

    printf("\n") ;

#endif


    hipFree(gpuWall);
    hipFree(gpuResult[0]);
    hipFree(gpuResult[1]);

    delete [] data;
    delete [] wall;
    delete [] result;

}

